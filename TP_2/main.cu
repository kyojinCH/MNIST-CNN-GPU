#include "cnn_layers.h"
#include <stdio.h>
#include <stdlib.h>

int main() {
    // Input Data Initialization
    float* raw_data = (float*)malloc(sizeof(float) * 32 * 32);
    float* C1_data = (float*)malloc(sizeof(float) * 6 * 28 * 28);
    float* S1_data = (float*)malloc(sizeof(float) * 6 * 14 * 14);
    float* C1_kernel = (float*)malloc(sizeof(float) * 6 * 5 * 5);

    Matrix2DInitRand(raw_data, 32, 32);
    Matrix3DInitZero(C1_data, 6, 28, 28);
    Matrix3DInitRand(C1_kernel, 6, 5, 5);

    float *d_raw, *d_C1, *d_kernel, *d_S1;
    hipMalloc(&d_raw, sizeof(float) * 32 * 32);
    hipMalloc(&d_C1, sizeof(float) * 6 * 28 * 28);
    hipMalloc(&d_kernel, sizeof(float) * 6 * 5 * 5);
    hipMalloc(&d_S1, sizeof(float) * 6 * 14 * 14);

    hipMemcpy(d_raw, raw_data, sizeof(float) * 32 * 32, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, C1_kernel, sizeof(float) * 6 * 5 * 5, hipMemcpyHostToDevice);

    cudaConv2D<<<28, 28>>>(d_raw, d_kernel, d_C1, 32, 32, 5, 6, 28, 28);
    cudaMeanPool<<<14, 14>>>(d_C1, d_S1, 28, 28, 6, 2, 14, 14);

    hipMemcpy(S1_data, d_S1, sizeof(float) * 6 * 14 * 14, hipMemcpyDeviceToHost);
    printf("Mean Pooling Output:\n");
    Matrix2DPrint(S1_data, 14, 14);

    hipFree(d_raw);
    hipFree(d_C1);
    hipFree(d_kernel);
    hipFree(d_S1);
    free(raw_data);
    free(C1_data);
    free(C1_kernel);
    free(S1_data);
    system("pause");
    return 0;
}

// nvcc main.cu cnn_layers.cu -o .\outputs\main -ccbin "C:\Program Files\Microsoft Visual Studio\2022\Community\VC\Tools\MSVC\14.42.34433\bin\Hostx64\x64"